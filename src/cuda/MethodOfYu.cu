#include "MethodOfYu.h"
#include "MethodOfYu_kernel.cu"
#include "Thrust.cuh"

MethodOfYu::MethodOfYu(GPUmemory *_gMemory, Params *_params)
{
    params = _params;
    gMemory = _gMemory;

	gMemory->SurfaceAlloFixedMem();
    constantMemCopy();

    int priority_high, priority_low;
	hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);

	hipStreamCreateWithPriority(&s1, hipStreamDefault, priority_high);
	hipStreamCreateWithPriority(&s2, hipStreamDefault, priority_low);
}

MethodOfYu::~MethodOfYu()
{
    hipStreamDestroy(s1);
	hipStreamDestroy(s2);

	checkMemUsed();

	gMemory->SurfaceFreeFixedMem();
	std::cout << "~~MethodOfYu" << std::endl;

	checkMemUsed();
}


void MethodOfYu::processingOfParticles()
{
    extractionOfSurfaceAndInvolveParticles();
    thrustscan_particles();
    memallocation_particles();
    streamcompact_particles();
    smoothedparticles();
    transformmatrices();
}


void MethodOfYu::processingOfVertices()
{
	extractionOfSurfaceVertices();
    thrustscan_vertices();
	memallocation_vertices();
    streamcompact_vertices();
}


void MethodOfYu::estimationOfscalarField()
{
	scalarfield();
}


void MethodOfYu::extractionOfSurfaceAndInvolveParticles()
{
    dim3 gridDim, blockDim;
	calcGridDimBlockDim(params->mGridParams.spSize, gridDim, blockDim); 

    estimationOfSurfaceParticles <<< gridDim, blockDim >>> (
		gMemory->dSpatialGrid,
		gMemory->dNumSurParticleGrid,
		gMemory->dNumInvParticleGrid,
		gMemory->dIndexRange);

	hipDeviceSynchronize();

	// estimationOfInvolveParticles  <<< gridDim, blockDim >>> (
	// 	gMemory->dNumSurParticleGrid,
	// 	gMemory->dNumInvParticleGrid);

	// hipDeviceSynchronize();
}

void MethodOfYu::thrustscan_particles()
{
    NumSurfaceParticles = ThrustExclusiveScan(
		gMemory->dNumSurParticleGridScan,
		gMemory->dNumSurParticleGrid,
		(uint)params->mGridParams.spSize);

	gMemory->NumSurfaceParticles = NumSurfaceParticles;

	if (NumSurfaceParticles == 0)
	{
		std::cerr << "No surface particle detected!\n";
		return;
	}

	std::cout << "mNumSurfaceParticles =  " << NumSurfaceParticles << std::endl;

	std::cout << "surface particles ratio: " << 
	static_cast<double>(NumSurfaceParticles)
		/ gMemory->NumParticles << std::endl;


	// NumInvolveParticles = ThrustExclusiveScan(
	// 	gMemory->dNumInvParticleGridScan,
	// 	gMemory->dNumInvParticleGrid,
	// 	(uint)params->mGridParams.spSize);

	// gMemory->NumInvolveParticles = NumInvolveParticles;


	// if (NumInvolveParticles == 0)
	// {
	// 	std::cerr << "No involve particle detected!\n";
	// 	return;
	// }

	// std::cout << "mNumInvolveParticles =  " << NumInvolveParticles << std::endl;

	// std::cout << "involve particles ratio: " << 
	// static_cast<double>(NumInvolveParticles)
	// 	/ gMemory->NumParticles << std::endl;
}


void MethodOfYu::streamcompact_particles()
{
    dim3 gridDim, blockDim;
	calcGridDimBlockDim(params->mGridParams.spSize, gridDim, blockDim);

	compactationOfParticles << < gridDim, blockDim, 0, s1>> > (
		gMemory->dNumSurParticleGrid,
		gMemory->dNumSurParticleGridScan,
		gMemory->dIndexRange,
		gMemory->dSurfaceParticlesIndex);	
	hipStreamSynchronize(s1);

	// compactationOfParticles << < gridDim, blockDim, 0,  s2>> > (
	// 	gMemory->dNumInvParticleGrid,
	// 	gMemory->dNumInvParticleGridScan,
	// 	gMemory->dIndexRange,
	// 	gMemory->dInvolveParticlesIndex);
	// hipStreamSynchronize(s2);
}

void MethodOfYu::smoothedparticles()
{
    dim3 gridDim, blockDim;
	calcGridDimBlockDim(NumSurfaceParticles, gridDim, blockDim);
	calculationOfSmoothed << < gridDim, blockDim , 0, s1>> > (
		gMemory->dFluidParticle,
		gMemory->dSurfaceParticlesMean,
		gMemory->dSurfaceParticlesSmoothed,
		gMemory->dIndexRange,
		gMemory->dSurfaceParticlesIndex,
		gMemory->dSpatialGrid);
	hipStreamSynchronize(s1);
	
	// calcGridDimBlockDim(NumInvolveParticles, gridDim, blockDim);
	// calculationOfSmoothedforInvovle << < gridDim, blockDim , 0, s2>> > (
	// 	gMemory->dFluidParticle,
	// 	gMemory->dInvolveParticlesSmoothed,
	// 	gMemory->dIndexRange,
	// 	gMemory->dInvolveParticlesIndex);
	// hipStreamSynchronize(s2);
}


void MethodOfYu::transformmatrices()
{
    dim3 gridDim, blockDim;
	calcGridDimBlockDim(NumSurfaceParticles, gridDim, blockDim);
	calculationOfTransformMatrices << < gridDim, blockDim, 0, s1 >> > (
		gMemory->dSurfaceParticlesMean,
		gMemory->dFluidParticle,
		gMemory->dIndexRange,
		gMemory->dSurfaceParticlesIndex,
		gMemory->dSVDMatrices);

	hipStreamSynchronize(s1);
}


void MethodOfYu::extractionOfSurfaceVertices()
{
	dim3 gridDim, blockDim;
	calcGridDimBlockDim(NumSurfaceParticles, gridDim, blockDim);
	estimationOfSurfaceVertices << < gridDim, blockDim>> > (
		gMemory->dFluidParticle,
		gMemory->dSurfaceParticlesSmoothed,
		gMemory->dIndexRange,
		gMemory->dSurfaceParticlesIndex,
		gMemory->dIsSurfaceVertices,
		gMemory->dSVDMatrices,
		gMemory->dSpatialGrid);

	hipDeviceSynchronize();
}

void MethodOfYu::thrustscan_vertices()
{
	NumSurfaceVertices = ThrustExclusiveScan(
		gMemory->dIsSurfaceVerticesScan,
		gMemory->dIsSurfaceVertices,
		(uint)params->mGridParams.scSize);

	gMemory->NumSurfaceVertices = NumSurfaceVertices;

	if (NumSurfaceVertices == 0)
	{
		std::cerr << "No surface vertex detected!\n";
		return;
	}

	std::cout << "mNumSurfaceVertices =  " << NumSurfaceVertices << std::endl;

	std::cout << "surface vertices ratio: " << static_cast<double>(NumSurfaceVertices) /
		(params->mGridParams.scSize) << std::endl;
}

void MethodOfYu::streamcompact_vertices()
{
	dim3 gridDim, blockDim;
	calcGridDimBlockDim(params->mGridParams.scSize, gridDim, blockDim);

	compactationOfSurfaceVertices << < gridDim, blockDim>> > (
		gMemory->dIsSurfaceVertices,
		gMemory->dIsSurfaceVerticesScan,
		gMemory->dSurfaceVerticesIndex);

	hipDeviceSynchronize();
}

void MethodOfYu::scalarfield()
{
	dim3 gridDim, blockDim;
	calcGridDimBlockDim(NumSurfaceVertices, gridDim, blockDim);

	computationOfScalarFieldGrid << < gridDim, blockDim>> > (
		gMemory->dFluidParticle,
		gMemory->dSurfaceParticlesSmoothed,
		gMemory->dInvolveParticlesSmoothed,
		gMemory->dIndexRange,
		gMemory->dSurfaceParticlesIndex,
		gMemory->dInvolveParticlesIndex,
		gMemory->dSurfaceVerticesIndex,
		gMemory->dNumSurParticleGrid,
		gMemory->dNumSurParticleGridScan,
		gMemory->dNumInvParticleGrid,
		gMemory->dNumInvParticleGridScan,
		gMemory->dSVDMatrices,
		gMemory->dScalarFiled);

	hipDeviceSynchronize();
}


void MethodOfYu::constantMemCopy()
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dSurfaceParams), &params->mSurfaceParams, sizeof(SurfaceParams)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dSimParams), &params->mSimParams, sizeof(SimParams)));

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dGridParams), &params->mGridParams, sizeof(GridParams)));
}


void MethodOfYu::constantMemSurAndInvPar_Num()
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dNumSurfaceParticles), 
	&gMemory->NumSurfaceParticles, sizeof(uint)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dNumInvolveParticles), 
	&gMemory->NumInvolveParticles, sizeof(uint)));	
}

void MethodOfYu::constantMemSurVer_Num()
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(dNumSurfaceVertices), 
	&gMemory->NumSurfaceVertices, sizeof(uint)));
}

void MethodOfYu::memallocation_particles()
{
    gMemory->memAllocation_particles();
	constantMemSurAndInvPar_Num();
}

void MethodOfYu::memallocation_vertices()
{
	gMemory->memAllocation_vertices();
	constantMemSurVer_Num();
}

